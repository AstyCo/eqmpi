#include "hip/hip_runtime.h"
#include "cuda.hpp"
#include "utils.hpp"

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#include <math.h>

#define LOAD_CONST_MEM(dest, src)\
    hipMemcpyToSymbol(HIP_SYMBOL(dest), &src, sizeof(dest), 0, hipMemcpyHostToDevice)

__constant__ static int N; /// size of 3D grid

__constant__ static int i_first;  /// first index of 3D grid
__constant__ static int j_first;  /// first index of 3D grid
__constant__ static int k_first;  /// first index of 3D grid

__constant__ static long i_count;  /// count in each dimension of 3D grid
__constant__ static long j_count;  /// count in each dimension of 3D grid
__constant__ static long k_count;  /// count in each dimension of 3D grid

__constant__ static real step_i; /// step in each dimension
__constant__ static real step_j; /// step in each dimension
__constant__ static real step_k; /// step in each dimension

__constant__ static real step_t; /// step time

__constant__ static real *arr;
__constant__ static real *arrP;
__constant__ static real *arrPP;

__constant__ static real sqrt3;

__device__
static real phi(real x, real y, real z)
{
    return sin(x) * cos(y - VAL_LY/2) * sin(z);
}

// Δ = div grad
// Δ phi = (d/dx^2 + d/dy^2 + d/dz^2) phi
__device__
static real div_grad_phi(real x, real y, real z)
{
    return -3.0 * phi(x, y, z);
}

__device__
static real u(real x, real y, real z, real t)
{
    return phi(x, y, z) * cos(sqrt3 * t);
}

__device__
static long get_index(uint i, uint j, uint k)
{
    return (long(i + 1) * (j_count + 2) + (j + 1)) * (k_count + 2) + (k + 1);
}

__device__
static real x_val(int i) { return (i_first + i) * step_i;}

__device__
static real y_val(int j) { return (j_first + j) * step_j;}

__device__
static real z_val(int k) { return (k_first + k) * step_k;}

struct Index
{
    int i, j, k;

    __device__
    Index(long id)
    {
        k = id % (k_count + 2);
        long ij = id / (k_count + 2);
        j = ij % (j_count + 2);
        i = ij / (j_count + 2);
    }
};

__device__
void calculate(long offset)
{
    Index id(offset);
    int i = id.i, j = id.j, k = id.k;

    arr[offset] = 2 * arrP[offset] - arrPP[offset]
            + step_t * step_t * (
                (arrP[get_index(i-1,j,k)]
                - 2 * arrP[offset]
                + arrP[get_index(i+1,j,k)]) / step_i / step_i
            + (arrP[get_index(i,j-1,k)]
            - 2 * arrP[offset]
            + arrP[get_index(i,j+1,k)]) / step_j / step_j
            + (arrP[get_index(i,j,k-1)]
            - 2 * arrP[offset]
            + arrP[get_index(i,j,k+1)]) / step_k / step_k
            );
}

void cuda_resize(RealDVector &dArray,
                 RealDVector &dArrayP,
                 RealDVector &dArrayPP,
                 RealDVector &dEdgeArray,
                 RealHVector &hEdgeArray,
                 RealDVector &dDeviationsArray,
                 long totalEdgeSize,
                 long bigsize)
{
    std::cout << "\tbig_size: " << bigsize << std::endl;
    std::cout << "\tmax real_d_vector_size: " << dArray.max_size() << std::endl;
    std::cout << "\tmax real_h_vector_size: " << hEdgeArray.max_size() << std::endl;

    dArray.resize(bigsize);
    dArrayP.resize(bigsize);
    dArrayPP.resize(bigsize);

    dEdgeArray.resize(totalEdgeSize);

    hEdgeArray.resize(totalEdgeSize);


    if (clargs.deviation)
        dDeviationsArray.resize(bigsize);

    std::cout << "allocation success" << std::endl;
}

struct FDeviation
{
    long size;
    real h_time;

    __host__
    FDeviation(long size_, real t_)
        : size(size_), h_time(t_)
    {}

    __device__
    real operator()(int offset) {
        Index id(offset);
        if (id.i == 0 || id.j == 0 || id.k == 0
                || id.i == i_count + 1
                || id.j == j_count + 1
                || id.k == k_count + 1) {
            return 0;
        }

        real val = arr[offset] - u(x_val(id.i), y_val(id.j), z_val(id.k), h_time);
        return ABS(val);
    }
};

real cuda_get_local_avg_deviation(long bigsize, long size, real current_time,
                                  RealDVector &dDeviationsArray)
{
    real result = 0;

    thrust::counting_iterator<int> first(0);

    thrust::transform(first, first + bigsize,
                      dDeviationsArray.begin(),
                      FDeviation(size, current_time));

    result = thrust::reduce(dDeviationsArray.begin(), dDeviationsArray.end(),
                            real(0),
                            thrust::plus<real>());

    result /= size;
    return result;
}

struct FStep0
{
    __device__
    real operator()(int offset) {
        Index id(offset);
        return phi(x_val(id.i), y_val(id.j), z_val(id.k));
    }
};

void cuda_step_0(RealDVector &dArray, RealDVector &dArrayPP)
{
    thrust::counting_iterator<int> first(0);
    thrust::transform(first, first + dArrayPP.size(),
                      dArrayPP.begin(), FStep0()); // install PHI

    dArray = dArrayPP; // install 0-type boundary conditions
}

struct FStep1
{
    __device__
    real operator()(int offset) {
        Index id(offset);
        return arrPP[offset]
                + step_t * step_t / 2 * div_grad_phi(x_val(id.i), y_val(id.j), z_val(id.k));
    }
};

void cuda_step_1(RealDVector &dArrayP)
{
    thrust::counting_iterator<int> first(0);
    thrust::transform(first, first + dArrayP.size(),
                      dArrayP.begin(), FStep1()); // install LAPLACIAN PHI
}

struct FCalculateInner
{
    __device__
    void operator()(int offset) {
        Index id(offset);
        if (id.i < 2 || id.j < 2 || id.k < 2
                || id.i > i_count - 1
                || id.j > j_count - 1
                || id.k > k_count - 1) {
            // don't change
        }
        else {
            calculate(offset);
        }
    }
};

void cuda_calculate_inner(long bigsize)
{
    thrust::counting_iterator<int> first(0);
    thrust::for_each_n(first, bigsize,
                       FCalculateInner()); // calculate inner val
}

struct FCalculateEdge
{
    __device__
    real operator()(int offset) {
        calculate(offset);
        return arr[offset];
    }
};

void cuda_calculate_edges(LongDVector &dEdgeIndices, RealDVector &dEdgeArray)
{
    thrust::transform(dEdgeIndices.begin(), dEdgeIndices.end(),
                      dEdgeArray.begin(), FCalculateEdge());
}

void cuda_shift_arrays(RealDVector &dArray,
                       RealDVector &dArrayP,
                       RealDVector &dArrayPP)
{
    dArrayPP = dArrayP;
    dArrayP = dArray;
}

void cuda_load_const_mem(int N_,
                         int i0_,int j0_, int k0_,
                         long ic_, long jc_, long kc_,
                         real hi_, real hj_, real hk_,
                         real ht_,
                         RealDVector *array_,
                         RealDVector *arrayP_,
                         RealDVector *arrayPP_)
{
    LOAD_CONST_MEM(N, N_);

    LOAD_CONST_MEM(i_first, i0_);
    LOAD_CONST_MEM(j_first, j0_);
    LOAD_CONST_MEM(k_first, k0_);

    LOAD_CONST_MEM(i_count, ic_);
    LOAD_CONST_MEM(j_count, jc_);
    LOAD_CONST_MEM(k_count, kc_);

    LOAD_CONST_MEM(step_i, hi_);
    LOAD_CONST_MEM(step_j, hj_);
    LOAD_CONST_MEM(step_k, hk_);

    LOAD_CONST_MEM(step_t, ht_);

    const real *arr_ = array_->data().get();
    const real *arrP_ = arrayP_->data().get();
    const real *arrPP_ = arrayPP_->data().get();
    LOAD_CONST_MEM(arr, arr_);
    LOAD_CONST_MEM(arrP, arrP_);
    LOAD_CONST_MEM(arrPP, arrPP_);

    real sqrt3_ = sqrt(3.0);
    LOAD_CONST_MEM(sqrt3, sqrt3_);
}


